#include "hip/hip_runtime.h"

#include "ProjHelperFun.cu.h"
#include "Constants.h"
#include "TridagPar.h"

#include "../include/CudaUtilProj.cu.h"

//#include "ProjHost.cu"

#define EPSILON 0.001
#define T 32

#define YX(k,j,i) ((k)*(numY)*(numX)+(j)*(numX)+(i))  //[-][numY][numX]
#define XY(k,j,i) ((k)*(numY)*(numX)+(j)*(numY)+(i)) //[-][numX][numY]
#define ZZ(k,j,i) (k*(numZ)*(numZ)+(j)*(numZ)+(i))    //[-][numZ][numZ]
#define D4ID(j,i) ((j)*4+(i))
#define X4(j,i) ((j)*numX+(i))
#define Y4(j,i) ((j)*numY+(i))


//{{{KERNELS  ------ 
__global__ void
d_initTimeline( REAL* d_timeline, const unsigned numT, const REAL t){
    unsigned gid = blockIdx.x*blockDim.x + threadIdx.x;
    if(gid < numT) {
        d_timeline[gid] =  t*gid / (numT-1);
    }
}


__global__ void
d_initNUM( REAL* d_num, unsigned int num_size, const REAL d, unsigned myIndex, const REAL s){
    const unsigned long gid = blockIdx.x*blockDim.x + threadIdx.x;
    if(gid < num_size) {
        d_num[gid] = gid*d - myIndex*d + s;
    }
}


__global__ void
d_initOperator( REAL* d_x, unsigned int x_size, REAL* d_dxx){
    const unsigned long gid = blockIdx.x*blockDim.x + threadIdx.x;
    if(gid < x_size) {
        REAL dxl, dxu;
        if(gid == 0){
            //  lower boundary
            dxl      =  0.0;
            dxu      =  d_x[1] - d_x[0];

            d_dxx[0] =  0.0;
            d_dxx[1] =  0.0;
            d_dxx[2] =  0.0;
            d_dxx[3] =  0.0;
        }else if(gid == x_size-1){
            //  upper boundary
            dxl        =  d_x[x_size-1] - d_x[x_size-2];
            dxu        =  0.0;

            d_dxx[(x_size-1)*4+0] = 0.0;
            d_dxx[(x_size-1)*4+1] = 0.0;
            d_dxx[(x_size-1)*4+2] = 0.0;
            d_dxx[(x_size-1)*4+3] = 0.0;
        }else{
            dxl      = d_x[gid]     - d_x[gid-1];
            dxu      = d_x[gid+1]   - d_x[gid];

            d_dxx[gid*4+0] =  2.0/dxl/(dxl+dxu);
            d_dxx[gid*4+1] = -2.0*(1.0/dxl + 1.0/dxu)/(dxl+dxu);
            d_dxx[gid*4+2] =  2.0/dxu/(dxl+dxu);
            d_dxx[gid*4+3] =  0.0;
        }
    }
}

__global__ void
d_setPayoff(REAL* d_result, REAL* d_x, unsigned int x_size, unsigned int y_size, unsigned int z_size){
    unsigned int x = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int y = blockDim.y*blockIdx.y + threadIdx.y;
    unsigned int z = blockDim.z*blockIdx.z + threadIdx.z;

    if(x < x_size && y < y_size && z < z_size){
        d_result[z*y_size*x_size + y*x_size + x] = max(d_x[y]-(0.001*z), (REAL)0.0);
    }
}


__global__ void
d_updateParams(REAL* d_varX, REAL* d_varY, REAL* d_x, REAL* d_y,  REAL* d_timeline,
    int g, REAL alpha, REAL beta, REAL nu, 
    unsigned int numX, unsigned int numY){

    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int j = blockDim.y*blockIdx.y + threadIdx.y;

    if(i >= numX || j >= numY)
        return;

    d_varX[i*numY+j] = exp(2.0*( beta*log(d_x[i]) +  d_y[j] - 0.5*nu*nu*d_timeline[g]));
    d_varY[i*numY+j] = exp(2.0*( alpha*log(d_x[i]) + d_y[j] - 0.5*nu*nu*d_timeline[g]));

}

__global__ void
d_updateParams_sh(REAL* d_varX, REAL* d_varY, REAL* d_x, REAL* d_y, REAL* d_timeline, 
    unsigned int g, REAL alpha, REAL beta, REAL nu, 
    unsigned int numX, unsigned int numY){

    __shared__ REAL sh_varX[T][T+1], sh_varY[T][T+1]; //

    __shared__ REAL sh_x[T], sh_y[T]; //

    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;  //numY
    unsigned int j = blockDim.y*blockIdx.y + threadIdx.y;  //numX
    int tidy = threadIdx.y;
    int tidx = threadIdx.x;

    if(j >= numX || i >= numY)
        return;

    // shared memory store operation
    sh_varX[tidy][tidx] = d_varX[j*numY+i];
    sh_varY[tidy][tidx] = d_varY[j*numY+i]; 
    sh_x[tidy] = d_x[j];
    sh_y[tidx] = d_y[i];
    __syncthreads();

    sh_varX[tidy][tidx] = exp(2.0*( beta*log(sh_x[tidy]) +  sh_y[tidx] - 0.5*nu*nu*d_timeline[g]));
    sh_varY[tidy][tidx] = exp(2.0*( alpha*log(sh_x[tidy]) + sh_y[tidx] - 0.5*nu*nu*d_timeline[g]));

    d_varX[j*numY+i] = sh_varX[tidy][tidx]; 
    d_varY[j*numY+i] = sh_varY[tidy][tidx]; 

}


__global__ void
d_explicit_xy_implicit_x(REAL* u, REAL* v, REAL* a, REAL* b, REAL* c,  
    REAL* varX, REAL* varY, REAL* timeline, REAL* dxx, REAL* dyy, REAL* result, 
    unsigned int g, unsigned numX, unsigned numY, unsigned outer, unsigned numZ){
   
    unsigned int k = blockDim.z * blockIdx.z + threadIdx.z; //Outer
    unsigned int j = blockDim.y * blockIdx.y + threadIdx.y; //numY
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; //numX


    if(k >= outer || j >= numY || i >= numX)
    return;
    
    //  explicit x 
    u[YX(k,j,i)] =  (1.0/(timeline[g+1]-timeline[g])) *result[XY(k,i,j)];

    if(i > 0) {
      u[YX(k,j,i)] += 0.5*( 0.5*varX[XY(0,i,j)]*dxx[D4ID(i,0)] ) 
            * result[XY(k,i-1,j)];
    }
    u[YX(k,j,i)]  +=  0.5*( 0.5*varX[XY(0,i,j)]*dxx[D4ID(i,1)] )
            * result[XY(k,i,j)];
    if(i < numX-1) {
      u[YX(k,j,i)]  +=  0.5*( 0.5*varX[XY(0,i,j)]*dxx[D4ID(i,2)] )
            * result[XY(k,i+1,j)];
    }

    //  explicit y ; RAW v, write u
    v[XY(k,i,j)] = 0.0;

    if(j > 0) {
      v[XY(k,i,j)] +=  ( 0.5*varY[XY(0,i,j)]*dyy[D4ID(j,0)] )
         *  result[XY(k,i,j-1)];
    }
      v[XY(k,i,j)] +=  ( 0.5*varY[XY(0,i,j)]*dyy[D4ID(j,1)] )
         *  result[XY(k,i,j)];
    if(j < numY-1) {
      v[XY(k,i,j)] +=  ( 0.5*varY[XY(0,i,j)]*dyy[D4ID(j,2)] )
         *  result[XY(k,i,j+1)];
    }
    u[YX(k,j,i)] += v[XY(k,i,j)];


    //  implicit x  // write a,b,c
    a[ZZ(k,j,i)] =       - 0.5*(0.5*varX[XY(0,i,j)]*dxx[D4ID(i,0)]);
    b[ZZ(k,j,i)] = ( 1.0/(timeline[g+1]-timeline[g])) - 0.5*(0.5*varX[XY(0,i,j)]*dxx[D4ID(i,1)]);
    c[ZZ(k,j,i)] =       - 0.5*(0.5*varX[XY(0,i,j)]*dxx[D4ID(i,2)]);
    
}



#define UI(k,j,i) ((k)*(middle)*(n)+(j)*(n)+(i))  

__global__ void
d_tridag_implicit_y(
    REAL* a, REAL* b, REAL* c, REAL* r, int n, REAL* u, REAL* uu, // tridag 
    unsigned numX, unsigned numY, unsigned outer, unsigned numZ, unsigned middle){

    unsigned int j = blockDim.x*blockIdx.x + threadIdx.x; //numX
    unsigned int k = blockDim.y*blockIdx.y + threadIdx.y; //outer

    if(k >= outer || j >= middle)
        return;
 
    REAL   beta;

    u[UI(k,j,0)]  = r[ZZ(k,j,0)]; //u[k*numX*numY + j*numY + i]
    uu[ZZ(k,j,0)] = b[ZZ(k,j,0)]; 

    for(int i=1; i< n; i++) {
        beta  = a[ZZ(k,j,i)] / uu[ZZ(k,j,i-1)];

        uu[ZZ(k,j,i)] = b[ZZ(k,j,i)] - beta*c[ZZ(k,j,i-1)];
        u[UI(k,j,i)]  = r[ZZ(k,j,i)] - beta*u[UI(k,j,i-1)];
    }

    u[UI(k,j,n-1)] = u[UI(k,j,n-1)] / uu[ZZ(k,j,n-1)];
    for(int i=n-2; i>=0; i--) {
        u[UI(k,j,i)]  = (u[UI(k,j,i)]  - c[ZZ(k,j,i)]*u[UI(k,j,i+1)] ) / uu[ZZ(k,j,i)];
    }
}

/*
__global__ void
sh_tridag_implicit_y(  // u = myresult
    REAL* a, REAL* b, REAL* c, REAL* r, int n, REAL* u, REAL* uu, // tridag 
    unsigned numX, unsigned numY, unsigned outer, unsigned numZ, unsigned middle){

    unsigned int j = blockDim.x*blockIdx.x + threadIdx.x; //numX
    unsigned int k = blockDim.y*blockIdx.y + threadIdx.y; //outer

    if(k >= outer || j >= middle)
        return;

    __shared__ REAL 
        // sh_a[T][T+1],   otherwise Too much shared data
        sh_b[T][T+1],
        sh_c[T][T+1], sh_r[T][T+1],
        sh_u[T][T+1], sh_uu[T][T+1]; //

    int tidy = threadIdx.y;
    int tidx = threadIdx.x;

    REAL   beta;

    for(int ii=0; ii< n; ii+=T) {

        // sh_a[tidy][tidx] = a[k*numZ*numZ +j*tidy + tidx];
        sh_b[tidy][tidx] = b[k*numZ*numZ +j*numZ + tidx];
        sh_c[tidy][tidx] = c[k*numZ*numZ +j*numZ + tidx];
        sh_r[tidy][tidx] = r[k*numZ*numZ +j*numZ + tidx];
        sh_uu[tidy][tidx] = uu[k*numZ*numZ +j*numZ + tidx];
        sh_u[tidy][tidx] = u[k* numX *numY +j*numZ + tidx]; // u and result are different!!
        
        __syncthreads();

        sh_u[tidy][0] = sh_r[tidy][0];
        sh_uu[tidy][0] = sh_b[tidy][0];
        __syncthreads();

        for(int i= 1; i< T; i++){
            // beta  = a[ZZ(k,j,i)] / uu[ZZ(k,j,i-1)];
            beta = a[ZZ(k,j,i)]  / sh_uu[tidy][i-1];
            __syncthreads();

            // uu[ZZ(k,j,i)] = b[ZZ(k,j,i)] - beta*c[ZZ(k,j,i-1)];
            sh_uu[tidy][i] = sh_b[tidy][i] - beta*sh_c[tidy][i-1];

            // u[UI(k,j,i)]  = r[ZZ(k,j,i)] - beta*u[UI(k,j,i-1)];
            sh_u[tidy][i] = sh_r[tidy][i] - beta*sh_u[tidy][i-1];
            __syncthreads();

        }

        // u[UI(k,j,n-1)] = u[UI(k,j,n-1)] / uu[ZZ(k,j,n-1)];
        sh_u[tidy][T-1] = sh_u[tidy][T-1]/ sh_uu[tidy][T-1];
        __syncthreads();

        // read c uu, write u
        for(int i=T-2; i>=0; i--) {  
            // u[UI(k,j,i)]  = (u[UI(k,j,i)]  - c[ZZ(k,j,i)]*u[UI(k,j,i+1)] ) / uu[ZZ(k,j,i)];
            sh_u[tidy][i] = (sh_u[tidy][i] - sh_c[tidy][i]*sh_u[tidy][i+1])/ sh_uu[tidy][tidx];
        }
        __syncthreads();

        // a[k*numZ*numZ +j*tidy + tidx]=sh_a[tidy][tidx] ;
        // b[k*numZ*numZ +j*tidy + tidx]=sh_b[tidy][tidx] ;
        // c[k*numZ*numZ +j*tidy + tidx]=sh_c[tidy][tidx] ;
        u[k*numZ*numZ +j*tidy + tidx]=sh_u[tidy][tidx] ;
        // uu[k*numZ*numZ+j*tidy +tidx] =sh_uu[tidy][tidx];

    }
}
*/

__global__ void
d_tridag_implicit_x(
    REAL* a, REAL* b, REAL* c, REAL* r, int n, REAL* u, REAL* uu, // tridag 
    unsigned numX, unsigned numY, unsigned outer, unsigned numZ, unsigned middle){

    unsigned int j = blockDim.x*blockIdx.x + threadIdx.x; //numY
    unsigned int k = blockDim.y*blockIdx.y + threadIdx.y; //outer

    if(k >= outer || j >= middle)
        return;
 
    REAL   beta;

    u[UI(k,j,0)]  = r[UI(k,j,0)];
    uu[ZZ(k,j,0)] = b[ZZ(k,j,0)]; //uu size?? [numZ][numZ]

    for(int i=1; i< n; i++) {
        beta  = a[ZZ(k,j,i)] / uu[ZZ(k,j,i-1)];

        uu[ZZ(k,j,i)] = b[ZZ(k,j,i)] - beta*c[ZZ(k,j,i-1)];
        u[UI(k,j,i)]  = r[UI(k,j,i)] - beta*u[UI(k,j,i-1)];
    }

    u[UI(k,j,n-1)] = u[UI(k,j,n-1)] / uu[ZZ(k,j,n-1)];
    for(int i=n-2; i>=0; i--) {
        u[UI(k,j,i)]  = (u[UI(k,j,i)]  - c[ZZ(k,j,i)]*u[UI(k,j,i+1)] ) / uu[ZZ(k,j,i)];
    }
}

__global__ void
d_implicit_y(REAL* u, REAL* v, REAL* a, REAL* b, REAL* c,  REAL* y,  
    REAL* varY, REAL* timeline, REAL* dyy, 
    unsigned int g, unsigned numX, unsigned numY, unsigned outer, unsigned numZ){
   
    unsigned int k = blockDim.z * blockIdx.z + threadIdx.z; //Outer
    unsigned int i = blockDim.y * blockIdx.y + threadIdx.y; //numX
    unsigned int j = blockDim.x * blockIdx.x + threadIdx.x; //numY


    if(k >= outer || j >= numY || i >= numX)
        return;

    a[ZZ(k,i,j)] =       - 0.5*(0.5*varY[XY(0,i,j)]*dyy[D4ID(j,0)]);
    b[ZZ(k,i,j)] = ( 1.0/(timeline[g+1]-timeline[g])) - 0.5*(0.5*varY[XY(0,i,j)]*dyy[D4ID(j,1)]);
    c[ZZ(k,i,j)] =       - 0.5*(0.5*varY[XY(0,i,j)]*dyy[D4ID(j,2)]);
    y[ZZ(k,i,j)] = ( 1.0/(timeline[g+1]-timeline[g])) * u[YX(k,j,i)] - 0.5*v[XY(k,i,j)];
}


__global__ void
d_implicit_y_trans(REAL* u_tr, REAL* v, REAL* a, REAL* b, REAL* c,  REAL* y,  
    REAL* varY, REAL* timeline, REAL* dyy_tr, 
    unsigned int g, unsigned numX, unsigned numY, unsigned outer, unsigned numZ){
   
    unsigned int k = blockDim.z * blockIdx.z + threadIdx.z; //Outer
    unsigned int i = blockDim.y * blockIdx.y + threadIdx.y; //numX
    unsigned int j = blockDim.x * blockIdx.x + threadIdx.x; //numY


    if(k >= outer || j >= numY || i >= numX)
        return;

    a[ZZ(k,i,j)] =       - 0.5*(0.5*varY[XY(0,i,j)]*dyy_tr[Y4(0,j)]);
    b[ZZ(k,i,j)] = ( 1.0/(timeline[g+1]-timeline[g])) - 0.5*(0.5*varY[XY(0,i,j)]*dyy_tr[Y4(1,j)]);
    c[ZZ(k,i,j)] =       - 0.5*(0.5*varY[XY(0,i,j)]*dyy_tr[Y4(2,j)]);
    y[ZZ(k,i,j)] = ( 1.0/(timeline[g+1]-timeline[g])) * u_tr[XY(k,i,j)] - 0.5*v[XY(k,i,j)];
}



__global__ void
sh_implicit_y(REAL* u_tr, REAL* v, REAL* a, REAL* b, REAL* c,  REAL* y,  
    REAL* varY, REAL* timeline, REAL* dyy_tr, 
    int g, unsigned numX, unsigned numY, unsigned outer, unsigned numZ){
   
    unsigned int k = blockDim.z * blockIdx.z + threadIdx.z; //Outer
    unsigned int i = blockDim.y * blockIdx.y + threadIdx.y; //numX
    unsigned int j = blockDim.x * blockIdx.x + threadIdx.x; //numY

    if(k >= outer || j >= numY || i >= numX)
        return;

    __shared__ REAL 
        sh_varY[T][T+1],  
        // sh_dyy[T][T+1],
        // sh_u[T][T+1],    
        // sh_v[T][T+1],
        sh_a[T][T+1], 
        sh_b[T][T+1], 
        sh_c[T][T+1], 
        sh_y[T][T+1]; //

    int tidy = threadIdx.y;
    int tidx = threadIdx.x;


    // copy data from global memory to shared memory
    // sh_u[tidy][tidx] = u_tr[XY(k,i,j)];
    sh_a[tidy][tidx] = a[ZZ(k,i,j)];
    sh_b[tidy][tidx] = b[ZZ(k,i,j)];
    sh_c[tidy][tidx] = c[ZZ(k,i,j)];
    // sh_v[tidy][tidx] = v[XY(k,i,j)] ;
    sh_varY[tidy][tidx] = varY[i*numY +j];
    sh_y[tidy][tidx] = y[ZZ(k,i,j)];
    // sh_dyy[tidy][tidx] = (i<4) ? dyy_tr[Y4(i,j)] : 0.0; // need transpose

    __syncthreads();

    // a[ZZ(k,i,j)] =       - 0.5*(0.5*varY[XY(0,i,j)]*dyy_tr[Y4(0,j)]);
    // b[ZZ(k,i,j)] = ( 1.0/(timeline[g+1]-timeline[g])) - 0.5*(0.5*varY[XY(0,i,j)]*dyy_tr[Y4(1,j)]);
    // c[ZZ(k,i,j)] =       - 0.5*(0.5*varY[XY(0,i,j)]*dyy_tr[Y4(2,j)]);
    // y[ZZ(k,i,j)] = ( 1.0/(timeline[g+1]-timeline[g])) * u[YX(k,j,i)] - 0.5*v[XY(k,i,j)];
    sh_a[tidy][tidx] = - 0.5*(0.5* sh_varY[tidy][tidx] * dyy_tr[Y4(0,j)]);
    sh_b[tidy][tidx] = ( 1.0/(timeline[g+1]-timeline[g])) - 0.5*(0.5*sh_varY[tidy][tidx]*dyy_tr[Y4(1,j)]);
    sh_c[tidy][tidx]  =       - 0.5*(0.5*sh_varY[tidy][tidx]*dyy_tr[Y4(2,j)]);
    sh_y[tidy][tidx] = ( 1.0/(timeline[g+1]-timeline[g])) * u_tr[XY(k,i,j)] - 0.5*v[XY(k,i,j)];

    a[ZZ(k,i,j)] = sh_a[tidy][tidx];
    a[ZZ(k,i,j)] = sh_b[tidy][tidx];
    a[ZZ(k,i,j)] = sh_c[tidy][tidx];
    y[ZZ(k,i,j)] = sh_y[tidy][tidx];
}




__global__ void sgmMatTranspose( REAL* A, REAL* trA, int rowsA, int colsA ){
    __shared__ REAL tile[T][T+1];
 
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
  
    unsigned int k = blockDim.z * blockIdx.z + threadIdx.z; //Outer
    unsigned int i = blockDim.y * blockIdx.y + threadIdx.y; //numX
    unsigned int j = blockDim.x * blockIdx.x + threadIdx.x; //numY
    
    A += k*rowsA*colsA; 
    trA += k*rowsA*colsA;
    
    if( j < colsA && i < rowsA )
        tile[tidy][tidx] = A[i* colsA + j];
    __syncthreads();
    
    i=blockIdx.y*blockDim.y+tidx; 
    j=blockIdx.x*blockDim.x+tidy;
    
    if( j < colsA && i < rowsA )
        trA[j*rowsA+i] = tile[tidx][tidy];
        // trA[XY(k,j,i)] = tile[tidx][tidy];
}



// 2D matrix transpose
__global__ void matTranspose2D(REAL* A, REAL* trA, int rowsA, int colsA){
    __shared__ REAL tile[T][T+1];

    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int j = blockIdx.x*T + tidx;
    int i = blockIdx.y*T + tidy;
    
    if( j < colsA && i < rowsA )
        tile[tidy][tidx] = A[i*colsA+j];
    __syncthreads();

    i = blockIdx.y*T + threadIdx.x;
    j = blockIdx.x*T + threadIdx.y;
    
    if( j < colsA && i < rowsA )
        trA[j*rowsA+i] = tile[tidx][tidy];
}



//{{{ wrapper 
void initGrid_GPU(  const REAL s0, const REAL alpha, const REAL nu,const REAL t, 
                    const unsigned numX, const unsigned numY, const unsigned numT, 
                    REAL* d_myX, REAL* d_myY, REAL* d_myTimeline, unsigned myXindex, 
                    unsigned myYindex) {

    const unsigned int BLOCK_SIZE = 256;
    unsigned int NUM_BLOCKS = ceil(numT / (float)BLOCK_SIZE);

    d_initTimeline<<<NUM_BLOCKS,BLOCK_SIZE>>>(d_myTimeline, numT, t);

    NUM_BLOCKS = ceil(numX / (float)BLOCK_SIZE);
    const REAL stdX = 20.0*alpha*s0*sqrt(t);
    const REAL dx = stdX/numX;
    d_initNUM<<<NUM_BLOCKS,BLOCK_SIZE>>>(d_myX, numX, dx, myXindex, s0);

    const REAL stdY = 10.0*nu*sqrt(t);
    const REAL dy = stdY/numY;
    const REAL logAlpha = log(alpha);
    NUM_BLOCKS = ceil(numY / (float)BLOCK_SIZE);
    d_initNUM<<<NUM_BLOCKS,BLOCK_SIZE>>>(d_myY, numY, dy, myYindex, logAlpha);
}

void initOperator_GPU(REAL* d_x, unsigned int x_size, REAL* d_dxx){
    const unsigned int BLOCK_SIZE = 256;
    unsigned int NUM_BLOCKS = ceil(x_size / (float)BLOCK_SIZE);

    d_initOperator<<<NUM_BLOCKS,BLOCK_SIZE>>>(d_x, x_size, d_dxx);
}



void   run_OrigCPU(  
                const unsigned int&   outer,
                const unsigned int&   numX,
                const unsigned int&   numY,
                const unsigned int&   numT,
                const REAL&           s0,
                const REAL&           t, 
                const REAL&           alpha, 
                const REAL&           nu, 
                const REAL&           beta,
                      REAL*           res   // [outer] RESULT
) {


//// ---------- GPU version -------------------- ////
    REAL *h_result; // final result

    // GPU variables
    REAL *d_x, *d_y, *d_timeline, *d_dxx, *d_dyy;
    REAL *d_result, *d_varX, *d_varY;
    REAL *d_a, *d_b, *d_c, *d_yy, *d_yyy, *d_u, *d_v;

    // myXindex myYindex are scalars
    const REAL stdX = 20.0*alpha*s0*sqrt(t);
    const REAL dx = stdX/numX;
    unsigned myXindex = static_cast<unsigned>(s0/dx) % numX;
    unsigned myYindex = static_cast<unsigned>(numY/2.0);

    unsigned numZ = max(numX,numY);

    int memsize_X = numX * sizeof(REAL);
    int memsize_Y = numY * sizeof(REAL);
    int memsize_T = numT * sizeof(REAL);
    int memsize_XY = numX * numY * sizeof(REAL);
    int memsize_OXY = outer * numX * numY * sizeof (REAL);
    int memsize_OZZ = outer * numZ * numZ * sizeof (REAL);

    // CPU variables
    h_result    = (REAL*) malloc (memsize_OXY);
   
    // GPU variables
    hipMalloc((void**)&d_result, memsize_OXY);  //[outer][numX][numY]
    hipMalloc((void**)&d_varX, memsize_XY); //[numX][numY]
    hipMalloc((void**)&d_varY, memsize_XY); //[numX][numY]
    hipMalloc((void**)&d_x, memsize_X); //[numX]
    hipMalloc((void**)&d_y, memsize_Y); //[numY]
    hipMalloc((void**)&d_timeline, memsize_T); //[numT]
    hipMalloc((void**)&d_dxx, 4 * memsize_X); //[numX][4]
    hipMalloc((void**)&d_dyy, 4 * memsize_Y); //[numY][4]

    //a b c yy yyy: [outer][numZ][numZ]
    hipMalloc((void**)&d_a , memsize_OZZ);
    hipMalloc((void**)&d_b , memsize_OZZ);
    hipMalloc((void**)&d_c , memsize_OZZ);
    hipMalloc((void**)&d_yy , memsize_OZZ); //y in seq code
    hipMalloc((void**)&d_yyy, memsize_OZZ); //yy in seq code
    hipMalloc((void**)&d_u , memsize_OXY); //d_u : [outer][numY][numX]
    hipMalloc((void**)&d_v , memsize_OXY); //d_v : [outer][numX][numY]

// for transpose 
    REAL * d_u_tr;
    REAL * d_dyy_tr;
    hipMalloc((void**)&d_u_tr , memsize_OXY); //d_u : [outer][numY][numX]
    hipMalloc((void**)&d_dyy_tr, memsize_Y *4);


//GPU init 
    initGrid_GPU(s0, alpha, nu,t, numX,numY, numT, d_x, d_y, d_timeline, myXindex, myYindex);
    initOperator_GPU( d_x, numX, d_dxx);
    initOperator_GPU( d_y, numY, d_dyy);


 // GPU setPayoff
    dim3 block_3D(32, 32, 1);
    dim3 grid_3D_OXY(ceil(numY/32.0), ceil(numX/32.0), ceil(outer/1.0));
    d_setPayoff<<<grid_3D_OXY, block_3D>>>(d_result, d_x, numY, numX, outer);
   
    dim3 block_2D(T,T);
    dim3 grid_2D_OX(ceil(numX/T), ceil((float)outer/T));
    dim3 grid_2D_OY(ceil(numY/T), ceil((float)outer/T));
    dim3 grid_2D_YX(ceil( numX / T ), ceil( numY / T ));   
    dim3 grid_3D_OYX(ceil(numX/32.0), ceil(numY/32.0),ceil(outer/1.0) );


// timeline loop
for(int g = numT-2;g>=0;--g) { // second outer loop, g


    //GPU updateParams  
    // d_updateParams<<< grid_2D_YX, block_2D >>>(d_varX, d_varY, d_x, d_y, d_timeline,g, 
    //      alpha, beta, nu, numX, numY);
    dim3 block_2D(T,T), grid_2D_XY(ceil( numY / T ),ceil( numX / T )); // sh
    d_updateParams_sh<<< grid_2D_XY, block_2D >>>(d_varX, d_varY, d_x, d_y, d_timeline,g, 
         alpha, beta, nu, numX, numY);
    
    
     // GPU rollback Part_1 
    d_explicit_xy_implicit_x<<<grid_3D_OYX, block_3D>>>(d_u,d_v,d_a,d_b,d_c,
        d_varX,d_varY,d_timeline,d_dxx,d_dyy,d_result, g, numX, numY, outer, numZ);


   // GPU rollback part-2  
    d_tridag_implicit_x <<< grid_2D_OY, block_2D >>> (d_a,d_b,d_c, d_u, numX,d_u,d_yyy,numX,numY,outer,numZ,numY);


   // GPU rollback part 3
    dim3 grid_2D_Y4(1, ceil((float)numY/T));
    matTranspose2D<<< grid_2D_Y4, block_2D >>>(d_dyy, d_dyy_tr, numY, 4);
    sgmMatTranspose <<< grid_3D_OYX, block_3D>>>( d_u, d_u_tr, numY, numX );

    d_implicit_y_trans<<< grid_3D_OXY, block_3D >>>(d_u_tr,d_v,d_a,d_b,d_c, d_yy,
        d_varY,d_timeline, d_dyy_tr, g, numX, numY, outer, numZ);


//----------/GPU rollback 4 
    // dim3 block_2D(T,T,1), grid_2D_OX(ceil(numX/T), ceil((float)outer/T), 1); // 3D kernel is also vaild
    d_tridag_implicit_y <<< grid_2D_OX, block_2D >>> (d_a,d_b,d_c,d_yy,numY,d_result,d_yyy,numX,numY,outer,numZ,numX);
    

} // Timeline loop end


    hipMemcpy( h_result         , d_result       , memsize_OXY        , hipMemcpyDeviceToHost);

    // read the final result
    #pragma omp parallel for default(shared) schedule(static) 
    for( unsigned  k = 0; k < outer; ++ k )  //outermost loop k
        res[k] = h_result[XY(k,myXindex,myYindex)];  //  tested OK

    hipFree(d_x); hipFree(d_y); hipFree(d_dxx);hipFree(d_dyy); hipFree(d_timeline); 
    hipFree(d_result); hipFree(d_varX); hipFree(d_varY);
    hipFree(d_a); hipFree(d_b);hipFree(d_c); hipFree(d_yy);hipFree(d_yyy); 
    hipFree(d_u); hipFree(d_v);
    
    free(h_result);
 //   #endif
}

