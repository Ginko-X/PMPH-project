#include "hip/hip_runtime.h"
#include "ProjHelperFun.cu.h"
#include "Constants.h"
#include "TridagPar.h"

#include "../include/CudaUtilProj.cu.h"

//#include "ProjHost.cu"

#define EPSILON 0.0001
#define T 32

//#define GPU_INIT_TEST // tested OK
//#define GPU_SETPAYOFF_TEST // tested OK
//#define GPU_UPDATE_PARAMS_TEST // tested OK
//#define GPU_ROLLBACK_PART_1_TEST // tested ok
#define GPU_ROLLBACK_PART_2_TEST


//{{{KERNELS  ------ 
__global__ void
d_initTimeline( REAL* d_timeline, const unsigned numT, const REAL t){
    unsigned gid = blockIdx.x*blockDim.x + threadIdx.x;
    if(gid < numT) {
        d_timeline[gid] =  t*gid / (numT-1);
    }
}


__global__ void
d_initNUM( REAL* d_num, unsigned int num_size, const REAL d, unsigned myIndex, const REAL s){
    const unsigned long gid = blockIdx.x*blockDim.x + threadIdx.x;
    if(gid < num_size) {
        d_num[gid] = gid*d - myIndex*d + s;
    }
}


__global__ void
d_initOperator( REAL* d_x, unsigned int x_size, REAL* d_dxx){
    const unsigned long gid = blockIdx.x*blockDim.x + threadIdx.x;
    if(gid < x_size) {
        REAL dxl, dxu;
        if(gid == 0){
            //  lower boundary
            dxl      =  0.0;
            dxu      =  d_x[1] - d_x[0];

            d_dxx[0] =  0.0;
            d_dxx[1] =  0.0;
            d_dxx[2] =  0.0;
            d_dxx[3] =  0.0;
        }else if(gid == x_size-1){
            //  upper boundary
            dxl        =  d_x[x_size-1] - d_x[x_size-2];
            dxu        =  0.0;

            d_dxx[(x_size-1)*4+0] = 0.0;
            d_dxx[(x_size-1)*4+1] = 0.0;
            d_dxx[(x_size-1)*4+2] = 0.0;
            d_dxx[(x_size-1)*4+3] = 0.0;
        }else{
            dxl      = d_x[gid]     - d_x[gid-1];
            dxu      = d_x[gid+1]   - d_x[gid];

            d_dxx[gid*4+0] =  2.0/dxl/(dxl+dxu);
            d_dxx[gid*4+1] = -2.0*(1.0/dxl + 1.0/dxu)/(dxl+dxu);
            d_dxx[gid*4+2] =  2.0/dxu/(dxl+dxu);
            d_dxx[gid*4+3] =  0.0;
        }
    }
}

__global__ void
d_setPayoff(REAL* d_result, REAL* d_x, unsigned int x_size, unsigned int y_size, unsigned int z_size){
    unsigned int x = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int y = blockDim.y*blockIdx.y + threadIdx.y;
    unsigned int z = blockDim.z*blockIdx.z + threadIdx.z;

    if(x < x_size && y < y_size && z < z_size){
        d_result[z*y_size*x_size + y*x_size + x] = max(d_x[y]-(0.001*z), (REAL)0.0);
    }
}


__global__ void
d_updateParams(REAL* d_varX, REAL* d_varY, REAL* d_x, REAL* d_y, REAL* d_timeline, 
    unsigned int g, REAL alpha, REAL beta, REAL nu, 
    unsigned int numX, unsigned int numY){

    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int j = blockDim.y*blockIdx.y + threadIdx.y;

    if(i >= numX || j >= numY)
        return;

    d_varX[i*numY+j] = exp(2.0*( beta*log(d_x[i]) +  d_y[j] - 0.5*nu*nu*d_timeline[g]));
    d_varY[i*numY+j] = exp(2.0*( alpha*log(d_x[i]) + d_y[j] - 0.5*nu*nu*d_timeline[g]));

}


#define YX(k,j,i) ((k)*(numY)*(numX)+(j)*(numX)+(i))
#define XY(k,j,i) ((k)*(numY)*(numX)+(j)*(numY)+(i))
#define ZZ(k,j,i) (k*(numZ)*(numZ)+(j)*(numZ)+(i))
#define D4ID(j,i) ((j)*4+(i))

__global__ void
d_explicit_xy_implicit_x(REAL* u, REAL* v, REAL* a, REAL* b, REAL* c,  
    REAL* varX, REAL* varY, REAL* timeline, REAL* dxx, REAL* dyy, REAL* result, 
    unsigned int g, unsigned numX, unsigned numY, unsigned outer, unsigned numZ){
   
    unsigned int k = blockDim.z * blockIdx.z + threadIdx.z; //Outer
    unsigned int j = blockDim.y * blockIdx.y + threadIdx.y; //numY
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; //numX


    if(k >= outer || j >= numY || i >= numX)
    return;
    
    //  explicit x 
    u[YX(k,j,i)] =  (1.0/(timeline[g+1]-timeline[g])) *result[XY(k,i,j)];

    if(i > 0) {
      u[YX(k,j,i)] += 0.5*( 0.5*varX[XY(0,i,j)]*dxx[D4ID(i,0)] ) 
            * result[XY(k,i-1,j)];
    }
    u[YX(k,j,i)]  +=  0.5*( 0.5*varX[XY(0,i,j)]*dxx[D4ID(i,1)] )
            * result[XY(k,i,j)];
    if(i < numX-1) {
      u[YX(k,j,i)]  +=  0.5*( 0.5*varX[XY(0,i,j)]*dxx[D4ID(i,2)] )
            * result[XY(k,i+1,j)];
    }

    //  explicit y ; RAW v, write u
    v[XY(k,i,j)] = 0.0;

    if(j > 0) {
      v[XY(k,i,j)] +=  ( 0.5*varY[XY(0,i,j)]*dyy[D4ID(j,0)] )
         *  result[XY(k,i,j-1)];
    }
      v[XY(k,i,j)] +=  ( 0.5*varY[XY(0,i,j)]*dyy[D4ID(j,1)] )
         *  result[XY(k,i,j)];
    if(j < numY-1) {
      v[XY(k,i,j)] +=  ( 0.5*varY[XY(0,i,j)]*dyy[D4ID(j,2)] )
         *  result[XY(k,i,j+1)];
    }
    u[YX(k,j,i)] += v[XY(k,i,j)];


    //  implicit x  // write a,b,c
    a[ZZ(k,j,i)] =       - 0.5*(0.5*varX[XY(0,i,j)]*dxx[D4ID(i,0)]);
    b[ZZ(k,j,i)] = ( 1.0/(timeline[g+1]-timeline[g])) - 0.5*(0.5*varX[XY(0,i,j)]*dxx[D4ID(i,1)]);
    c[ZZ(k,j,i)] =       - 0.5*(0.5*varX[XY(0,i,j)]*dxx[D4ID(i,2)]);
    
}



__global__ void
d_implicit_x_tridag(
    REAL* a, REAL* b, REAL* c, REAL* r, int n, REAL* u, REAL* uu, // tridag 
    unsigned numX, unsigned numY, unsigned outer, unsigned numZ){

    unsigned int k = blockDim.x*blockIdx.x + threadIdx.x;
    unsigned int j = blockDim.y*blockIdx.y + threadIdx.y;

    if(k >= outer || j >= numY)
        return;

    // tridagPar(a[k][j],b[k][j],c[k][j],u[k][j],numX,u[k][j],yy[k][j]);  
    REAL   beta;

    u[YX(k,j,0)]  = r[YX(k,j,0)];
    uu[ZZ(k,j,0)] = b[ZZ(k,j,0)]; //uu size?? [numZ][numZ]

    for(int i=1; i< n; i++) {
        beta  = a[ZZ(k,j,i)] / uu[ZZ(k,j,i-1)];

        uu[ZZ(k,j,i)] = b[ZZ(k,j,i)] - beta*c[ZZ(k,j,i-1)];
        u[YX(k,j,i)]  = r[YX(k,j,i)] - beta*u[YX(k,j,i-1)];
    }

    u[ZZ(k,j,n-1)] = u[ZZ(k,j,n-1)] / uu[ZZ(k,j,n-1)];
    for(int i=n-2; i>=0; i--) {
        u[YX(k,j,i)]  = (u[YX(k,j,i)]  - c[ZZ(k,j,i)]*u[YX(k,j,i+1)] ) / uu[ZZ(k,j,i)];
    }
}


__global__ void
d_implicit_y(REAL* u, REAL* v, REAL* a, REAL* b, REAL* c,  REAL* y,  
    REAL* varY, REAL* timeline, REAL* dyy, 
    unsigned int g, unsigned numX, unsigned numY, unsigned outer, unsigned numZ){
   
    unsigned int k = blockDim.z * blockIdx.z + threadIdx.z; //Outer
    unsigned int i = blockDim.y * blockIdx.y + threadIdx.y; //numX
    unsigned int j = blockDim.x * blockIdx.x + threadIdx.x; //numY


    if(k >= outer || j >= numY || i >= numX)
    return;

    a[ZZ(k,i,j)] =       - 0.5*(0.5*varY[XY(0,i,j)]*dyy[D4ID(j,0)]);
    b[ZZ(k,i,j)] = ( 1.0/(timeline[g+1]-timeline[g])) - 0.5*(0.5*varY[XY(0,i,j)]*dyy[D4ID(j,1)]);
    c[ZZ(k,i,j)] =       - 0.5*(0.5*varY[XY(0,i,j)]*dyy[D4ID(j,2)]);
    y[ZZ(k,i,j)] = ( 1.0/(timeline[g+1]-timeline[g])) * u[YX(k,j,i)] - 0.5*v[XY(k,i,j)];
}



//{{{ wrapper 
void initGrid_GPU(  const REAL s0, const REAL alpha, const REAL nu,const REAL t, 
                    const unsigned numX, const unsigned numY, const unsigned numT, 
                    REAL* d_myX, REAL* d_myY, REAL* d_myTimeline, unsigned myXindex, 
                    unsigned myYindex) {

    const unsigned int BLOCK_SIZE = 256;
    unsigned int NUM_BLOCKS = ceil(numT / (float)BLOCK_SIZE);

    d_initTimeline<<<NUM_BLOCKS,BLOCK_SIZE>>>(d_myTimeline, numT, t);

    NUM_BLOCKS = ceil(numX / (float)BLOCK_SIZE);
    const REAL stdX = 20.0*alpha*s0*sqrt(t);
    const REAL dx = stdX/numX;
    d_initNUM<<<NUM_BLOCKS,BLOCK_SIZE>>>(d_myX, numX, dx, myXindex, s0);

    const REAL stdY = 10.0*nu*sqrt(t);
    const REAL dy = stdY/numY;
    const REAL logAlpha = log(alpha);
    NUM_BLOCKS = ceil(numY / (float)BLOCK_SIZE);
    d_initNUM<<<NUM_BLOCKS,BLOCK_SIZE>>>(d_myY, numY, dy, myYindex, logAlpha);
}

void initOperator_GPU(REAL* d_x, unsigned int x_size, REAL* d_dxx){
    const unsigned int BLOCK_SIZE = 256;
    unsigned int NUM_BLOCKS = ceil(x_size / (float)BLOCK_SIZE);

    d_initOperator<<<NUM_BLOCKS,BLOCK_SIZE>>>(d_x, x_size, d_dxx);
}




// read a b c r, write u
inline void tridag(
    const vector<REAL>&   a,   // size [n]
    const vector<REAL>&   b,   // size [n]
    const vector<REAL>&   c,   // size [n]
    const vector<REAL>&   r,   // size [n]
    const int             n,
          vector<REAL>&   u,   // size [n]
          vector<REAL>&   uu   // size [n] temporary
) {
    int    i; 
    // int offset;
    REAL   beta;

    u[0]  = r[0];
    uu[0] = b[0];

    for(i=1; i<n; i++) {
        beta  = a[i] / uu[i-1];

        uu[i] = b[i] - beta*c[i-1];
        u[i]  = r[i] - beta*u[i-1];
    }

#if 1
    // X) this is a backward recurrence
    u[n-1] = u[n-1] / uu[n-1];
    for(i=n-2; i>=0; i--) {
        u[i] = (u[i] - c[i]*u[i+1]) / uu[i];
    }
#else
    // Hint: X) can be written smth like (once you make a non-constant)
    for(i=0; i<n; i++) a[i] =  u[n-1-i];
    a[0] = a[0] / uu[n-1];
    for(i=1; i<n; i++) a[i] = (a[i] - c[n-1-i]*a[i-1]) / uu[n-1-i];
    for(i=0; i<n; i++) u[i] = a[n-1-i];
#endif
}



void   run_OrigCPU(  
                const unsigned int&   outer,
                const unsigned int&   numX,
                const unsigned int&   numY,
                const unsigned int&   numT,
                const REAL&           s0,
                const REAL&           t, 
                const REAL&           alpha, 
                const REAL&           nu, 
                const REAL&           beta,
                      REAL*           res   // [outer] RESULT
) {

    PrivGlobs  globs(numX, numY, numT);
 
    initGrid    (s0,alpha,nu,t, numX, numY, numT, globs);
    initOperator(globs.myX,globs.myDxx);
    initOperator(globs.myY,globs.myDyy);


    // array expansion on myResult (originally globs.myResult) from [numX][numY] to [outer][numX][numY]
    vector<vector<vector<REAL> > > myResult;
    myResult.resize(outer); 
#pragma omp parallel for default(shared) schedule(static)    
    for(int i=0; i<outer; i++) {
        myResult[i].resize(numX);
        for(int j=0; j<numX; j++){
            myResult[i][j].resize(numY);
       }
    }

    //myVarX myVarY: [numX][numY]
    vector<vector<REAL> > myVarX, myVarY;     
    myVarX.resize(numX);
    myVarY.resize(numX);
    for(int i=0; i<numX; i++){
        myVarX[i].resize(numY);
        myVarY[i].resize(numY);
    }


unsigned numZ = max(numX, numY);


// array expansion on a, b, c, y, yy, [outer][numZ][numZ]
vector<vector<vector<REAL> > > a,b,c,y,yy;
a.resize(outer);
b.resize(outer);
c.resize(outer);
y.resize(outer);
yy.resize(outer);

#pragma omp parallel for default(shared) schedule(static)    
for(int i=0; i<outer; i++) {
    a[i].resize(numZ);
    b[i].resize(numZ);
    c[i].resize(numZ);
    y[i].resize(numZ);
    yy[i].resize(numZ);

    for(int j=0; j<numZ; j++){
       a[i][j].resize(numZ);
       b[i][j].resize(numZ);
       c[i][j].resize(numZ);
       y[i][j].resize(numZ);
       yy[i][j].resize(numZ);
   }
}
 
// array expansion on u,v, u is [outer][numY][numX], v is [outer][numX][]
vector<vector<vector<REAL> > > u,v;
u.resize(outer);
v.resize(outer);

for(int k=0; k<outer; k++){
    u[k].resize(numY);
    for(int i=0; i< numY; i++)
        u[k][i].resize(numX);

    v[k].resize(numX);
    for(int i=0; i< numX; i++)
        v[k][i].resize(numY);
}


      
  // setPayoff(strike, globs);  it's parallel so can be loop-distributed on the outmost loop
  // also need to do array expansion on globs.myResult, i.e.  myResult
#pragma omp parallel for default(shared) schedule(static)  //Kernel-1: 3D
    for( unsigned k = 0; k < outer; ++ k ) {  // outmost loop
        
        // modified setPayoff function below
        for(unsigned i=0;i<globs.myX.size();++i)
        {
            //REAL payoff = max(globs.myX[i]-strike, (REAL)0.0); // move this inside the loop to do privatization
            for(unsigned j=0;j<globs.myY.size();++j) 
                // globs.myResult[i][j] = payoff;   // note that payoff is just a scalar variables,
                myResult[k][i][j] = max(globs.myX[i]-(0.001*k), (REAL)0.0); 
        }
    }
        
  
//--- original code: 
// for(int i = globs.myTimeline.size()-2;i>=0;--i)
//     {
//         updateParams(i,alpha,beta,nu,globs);
//         rollback(i, globs);
//     }
//--- use loop interchange and loop distribution


//modified updateParams(g,alpha,beta,nu,globs);
  // Kernel-2: 3D
    for(int g = globs.myTimeline.size()-2;g>=0;--g) { // second outer loop, g

        #pragma omp parallel for default(shared) schedule(static)  // Kernel-2: 2D
        for(unsigned i=0;i<globs.myX.size();++i){
            for(unsigned j=0;j<globs.myY.size();++j) {
                myVarX[i][j] = exp(2.0*(  beta*log(globs.myX[i])   
                                      + globs.myY[j]             
                                      - 0.5*nu*nu*globs.myTimeline[g] )
                                );
                myVarY[i][j] = exp(2.0*(  alpha*log(globs.myX[i])   
                                      + globs.myY[j]             
                                      - 0.5*nu*nu*globs.myTimeline[g] )
                               ); // nu*nu
            }
        }
        
        // rollback Part 1, write u,v, a, b, c  
        #pragma omp parallel for default(shared) schedule(static)   // Kernel-3: 3D
        for( unsigned k = 0; k < outer; ++ k ) {  //outermost loop k, after interchanged //Par        
            for(unsigned j=0;j<numY;j++) {  // interchanged with the inner loop
                for(unsigned i=0;i<numX;i++) {

                    //  explicit x 
                    u[k][j][i] =  (1.0/(globs.myTimeline[g+1]-globs.myTimeline[g])) *myResult[k][i][j];

                    if(i > 0) { 
                      u[k][j][i] += 0.5*( 0.5*myVarX[i][j]*globs.myDxx[i][0] ) 
                                    * myResult[k][i-1][j];
                    }
                    u[k][j][i]  +=  0.5*( 0.5*myVarX[i][j]*globs.myDxx[i][1] )
                                    * myResult[k][i][j];
                    if(i < numX-1) {
                      u[k][j][i] += 0.5*( 0.5*myVarX[i][j]*globs.myDxx[i][2] )
                                    * myResult[k][i+1][j];
                    }

                    //  explicit y ; RAW v, write u
                    v[k][i][j] = 0.0;

                    if(j > 0) {
                      v[k][i][j] +=  ( 0.5*myVarY[i][j]*globs.myDyy[j][0] )
                                 *  myResult[k][i][j-1];
                    }
                    v[k][i][j]  +=   ( 0.5*myVarY[i][j]*globs.myDyy[j][1] )
                                 *  myResult[k][i][j];
                    if(j < numY-1) {
                      v[k][i][j] +=  ( 0.5*myVarY[i][j]*globs.myDyy[j][2] )
                                 *  myResult[k][i][j+1];
                    }
                    u[k][j][i] += v[k][i][j]; 

                    
                    //  implicit x  // write a,b,c
                    a[k][j][i] =       - 0.5*(0.5*myVarX[i][j]*globs.myDxx[i][0]);
                    b[k][j][i] = ( 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g])) - 0.5*(0.5*myVarX[i][j]*globs.myDxx[i][1]);
                    c[k][j][i] =       - 0.5*(0.5*myVarX[i][j]*globs.myDxx[i][2]);
                }
            }
        }
        
        //Part 2 : read a,b,c,u to write u
    #pragma omp parallel for default(shared) schedule(static)  //kernel-4: 2D Kernel or can be merged with the last one to make a 2D kernel
        for( unsigned k = 0; k < outer; ++ k ) {  //outermost loop distribution //Par
            for(unsigned j=0;j<numY;j++) {  // Par
                tridagPar(a[k][j],b[k][j],c[k][j],u[k][j],numX,u[k][j],yy[k][j]);  
            }
        }

        //Part 3, write a b c y reading from u,v    // implicit y, 
    #pragma omp parallel for default(shared) schedule(static)  // Kernel-5: 3D
        for( unsigned k = 0; k < outer; ++ k ) {  //outermost loop distribution //Par
            for(unsigned i=0;i<numX;i++) { 
                for(unsigned j=0;j<numY;j++) {  
                    a[k][i][j] =       - 0.5*(0.5*myVarY[i][j]*globs.myDyy[j][0]);
                    b[k][i][j] = ( 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g])) - 0.5*(0.5*myVarY[i][j]*globs.myDyy[j][1]);
                    c[k][i][j] =       - 0.5*(0.5*myVarY[i][j]*globs.myDyy[j][2]);
               
                    y[k][i][j] = ( 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g])) *u[k][j][i] - 0.5*v[k][i][j];
                }
            }
        }

        //Part 4: write myResult reading from a b c y 
    #pragma omp parallel for default(shared) schedule(static)   //kernel-6  
        for( unsigned k = 0; k < outer; ++ k ) {  //outermost loop distribution //Par
            for(unsigned i=0;i<numX;i++) { 
                tridagPar(a[k][i],b[k][i],c[k][i],y[k][i],numY,myResult[k][i],yy[k][i]);
            }
        }


    }



#pragma omp parallel for default(shared) schedule(static) 
for( unsigned  k = 0; k < outer; ++ k )  //outermost loop k
    res[k] = myResult[k][globs.myXindex][globs.myYindex]; // myRes[0][k];



//// ---------- GPU version -------------------- ////
// globs vars for gpu
    REAL *h_result; // the final result

    // GPU variables
    REAL *d_x, *d_y, *d_timeline, *d_dxx, *d_dyy;
    REAL *d_result, *d_varX, *d_varY;
    REAL *d_a, *d_b, *d_c, *d_yy, *d_yyy, *d_u, *d_v;

    // myXindex myYindex are scalars
    const REAL stdX = 20.0*alpha*s0*sqrt(t);
    const REAL dx = stdX/numX;
    unsigned myXindex = static_cast<unsigned>(s0/dx) % numX;
    // const REAL stdY = 10.0*nu*sqrt(t);
    // const REAL dy = stdY/numY;
    // const REAL logAlpha = log(alpha);
    unsigned myYindex = static_cast<unsigned>(numY/2.0);


    int memsize_X = numX * sizeof(REAL);
    int memsize_Y = numY * sizeof(REAL);
    int memsize_T = numT * sizeof(REAL);
    int memsize_XY = numX * numY * sizeof(REAL);
    int memsize_OXY = outer * numX * numY * sizeof (REAL);
    int memsize_OZZ = outer * numZ * numZ * sizeof (REAL);

    // CPU variables
    h_result    = (REAL*) malloc (memsize_OXY);

    // GPU variables
    hipMalloc((void**)&d_result, memsize_OXY);  //[outer][numX][numY]
    hipMalloc((void**)&d_varX, memsize_XY); //[numX][numY]
    hipMalloc((void**)&d_varY, memsize_XY); //[numX][numY]
    hipMalloc((void**)&d_x, memsize_X); //[numX]
    hipMalloc((void**)&d_y, memsize_Y); //[numY]
    hipMalloc((void**)&d_timeline, memsize_T); //[numT]
    hipMalloc((void**)&d_dxx, 4 * memsize_X); //[numX][4]
    hipMalloc((void**)&d_dyy, 4 * memsize_Y); //[numY][4]

    //a b c yy yyy: [outer][numZ][numZ]
    hipMalloc((void**)&d_a , memsize_OZZ);
    hipMalloc((void**)&d_b , memsize_OZZ);
    hipMalloc((void**)&d_c , memsize_OZZ);
    hipMalloc((void**)&d_yy , memsize_OZZ); //y in seq code
    hipMalloc((void**)&d_yyy, memsize_OZZ); //yy in seq code

    hipMalloc((void**)&d_u , memsize_OXY); //d_u : [outer][numY][numX]
    hipMalloc((void**)&d_v , memsize_OXY); //d_v : [outer][numX][numY]

//GPU init 

    initGrid_GPU(s0, alpha, nu,t, numX,numY, numT, d_x, d_y, d_timeline, myXindex, myYindex);
    initOperator_GPU( d_x, numX, d_dxx);
    initOperator_GPU( d_y, numY, d_dyy);


 // test GPU init
 #ifdef GPU_INIT_TEST      
    REAL *h_x, *h_y, *h_timeline, *h_dxx, *h_dyy;
    h_x         = (REAL *) malloc (memsize_X          );
    h_y         = (REAL *) malloc (memsize_Y          );
    h_timeline  = (REAL *) malloc (memsize_T          );
    h_dxx       = (REAL *) malloc (4* memsize_X       );
    h_dyy       = (REAL *) malloc (4* memsize_Y       );
    hipMemcpy( h_x         , d_x       , numX*sizeof(REAL)           , hipMemcpyDeviceToHost);
    hipMemcpy( h_y         , d_y       , numY*sizeof(REAL)           , hipMemcpyDeviceToHost);
    hipMemcpy( h_timeline  , d_timeline, memsize_T                   , hipMemcpyDeviceToHost);
    hipMemcpy( h_dxx       , d_dxx     , numX*4*sizeof(REAL)         , hipMemcpyDeviceToHost);
    hipMemcpy( h_dyy       , d_dyy     , numY*4*sizeof(REAL)         , hipMemcpyDeviceToHost);


        bool valid = true;
        for(int i = 0; i < numX; i++){
            if(abs(h_x[i]-globs.myX[i]) > EPSILON){
                valid = false;
                printf("\n** invalid h_x  %f  %f**\n",
                          h_x[i], globs.myX[i]);
                break;
            }
        }

        for(int i = 0; i < numY; i++){
            if(abs(h_y[i]-globs.myY[i]) > EPSILON){
                valid = false;
                printf("\n** invalid h_y **\n");
                break;
            }
        }

        for(int i = 0; i < numT; i++){
            if(abs(h_timeline[i]-globs.myTimeline[i]) > EPSILON){
                valid = false;
                 printf("\n** invalid h_timeline  %d  %d**\n",
                          h_timeline[i], globs.myTimeline[i]);             
                break;
            }
        }
        for(int i = 0; i < numX*4; i++){
            if(abs(h_dxx[i]-globs.myDxx[i/4][i%4]) > EPSILON){
                valid = false;
                printf("\n** Invalid h_dxx **\n");                
                break;
            }
        }
        for(int i = 0; i < numY*4; i++){
            if(abs(h_dyy[i]-globs.myDyy[i/4][i%4]) > EPSILON){
                valid = false;
                printf("\n**  Invalid h_dyy **\n");            
                break;
            }
        }
        if(!valid){
            printf("\n**Initialization did not validate**\n");
        }

        free(h_x);free(h_y);free(h_timeline);free(h_dxx);free(h_dyy);
#endif

// GPU setPayoff
    dim3 block_3D(8, 8, 8);
    dim3 grid_3D_OXY(ceil(numY/8.0), ceil(numX/8.0), ceil(outer/8.0));
    d_setPayoff<<<grid_3D_OXY, block_3D>>>(d_result, d_x, numY, numX, outer);
  
    
#ifdef GPU_SETPAYOFF_TEST
    hipMemcpy( h_result    , d_result       , memsize_OXY, hipMemcpyDeviceToHost);

    for(int k = 0; k < outer; k++)
        for(int i = 0; i < globs.myX.size(); i++)
            for(int j = 0; j < globs.myY.size(); j++){
                REAL myResultTemp = max(globs.myX[i]-(0.001*k), (REAL)0.0); 
                if(abs(h_result[k*numX*numY+i*numY+j]-myResultTemp) > EPSILON){
                    printf("\n**SetPayOff did not validate %f  %f**\n",
                            h_result[k*numX*numY+i*numY+j], myResultTemp);
                    break;
                }
            }
#endif


//GPU updateParams
    int dimy = ceil( numY / T );
    int dimx = ceil( numX / T );
    dim3 block(T,T), grid(dimx,dimy);
     
    // Test only when g = 0， the last Timeline iteration
    d_updateParams<<< grid, block >>>(d_varX, d_varY, d_x, d_y, d_timeline, 0, alpha, beta, nu, numX, numY);
    

#ifdef GPU_UPDATE_PARAMS_TEST
    // Test only when g = 0， the last Timeline iteration, i.e.,
    // d_updateParams<<< grid, block >>>(d_varX, d_varY, d_x, d_y, d_timeline, 0, alpha, beta, nu, numX, numY);

    REAL *h_varX, *h_varY;  
    h_varX      = (REAL*) malloc (memsize_XY     );
    h_varY      = (REAL*) malloc (memsize_XY     );

    hipMemcpy( h_varX      , d_varX       , memsize_XY    , hipMemcpyDeviceToHost);
    hipMemcpy( h_varY      , d_varY    , memsize_XY     , hipMemcpyDeviceToHost);

    for(int i = 0; i < numX*numY; i++){
     if(abs(h_varX[i] - myVarX[i/numY][i%numY]) > EPSILON || abs(h_varY[i] - myVarY[i/numY][i%numY]) > EPSILON){
         printf("\n**Update Params did not validate %f=%f and %f=%f**\n",
                h_varX[i], myVarX[i/numY][i%numY], h_varY[i], myVarY[i/numY][i%numY]);
         break;
      }
    }

    free(h_varX); free(h_varY);
#endif 

    // GPU rollback Part_1  
    // const dim3 block_3D(8, 8, 8);
    const dim3 grid_3D_OYX(ceil(numX/8.0), ceil(numY/8.0),ceil(outer/8.0) );
    d_explicit_xy_implicit_x<<<grid_3D_OYX, block_3D>>>(d_u,d_v,d_a,d_b,d_c,
        d_varX,d_varY,d_timeline,d_dxx,d_dyy,d_result, 
        0, // firstly only test the case that g = 0
        numX, numY, outer, numZ);

    dimx = ceil( outer / T );
    dimy = ceil( numY / T );
    dim3 block_2D_OY(T,T), grid_2D_OY(dimx,dimy);
     
    d_implicit_x_tridag<<< grid_2D_OY, block_2D_OY >>>(d_a, d_b,d_c,d_u, numX, d_u, d_yy, numX, numY, outer, numZ); 


#ifdef GPU_ROLLBACK_PART_1_TEST      
    REAL *h_a, *h_b, *h_c, 
    // *h_yy, *h_yyy, 
    *h_u, *h_v;

    h_a         = (REAL *) malloc (memsize_OZZ         );
    h_b         = (REAL *) malloc (memsize_OZZ          );
    h_c         = (REAL *) malloc (memsize_OZZ          );
    // h_yy        = (REAL *) malloc (memsize_OZZ      );
    // h_yyy       = (REAL *) malloc (memsize_OZZ       );
    h_u        = (REAL *) malloc (memsize_OXY      );
    h_v       = (REAL *) malloc (memsize_OXY       );

    hipMemcpy( h_a         , d_a       , memsize_OZZ        , hipMemcpyDeviceToHost);
    hipMemcpy( h_b         , d_b       , memsize_OZZ        , hipMemcpyDeviceToHost);
    hipMemcpy( h_c         , d_c       , memsize_OZZ        , hipMemcpyDeviceToHost);
    // hipMemcpy( h_yy        , d_yy      , memsize_OZZ        , hipMemcpyDeviceToHost);
    // hipMemcpy( h_yyy       , d_yyy     , memsize_OZZ        , hipMemcpyDeviceToHost);
    hipMemcpy( h_u         , d_u       , memsize_OXY        , hipMemcpyDeviceToHost);
    hipMemcpy( h_v         , d_v       , memsize_OXY        , hipMemcpyDeviceToHost);


    unsigned g = 0;
    #pragma omp parallel for default(shared) schedule(static)   // Kernel-3: 3D
      for( unsigned k = 0; k < outer; ++ k ) {  //outermost loop k, after interchanged //Par        
        for(unsigned j=0;j<numY;j++) {  // interchanged with the inner loop
            for(unsigned i=0;i<numX;i++) {

                 // explicit x 
                u[k][j][i] =  (1.0/(globs.myTimeline[g+1]-globs.myTimeline[g])) *myResult[k][i][j];

                if(i > 0) { 
                  u[k][j][i] += 0.5*( 0.5*myVarX[i][j]*globs.myDxx[i][0] ) 
                                * myResult[k][i-1][j];
                }
                u[k][j][i]  +=  0.5*( 0.5*myVarX[i][j]*globs.myDxx[i][1] )
                                * myResult[k][i][j];
                if(i < numX-1) {
                  u[k][j][i] += 0.5*( 0.5*myVarX[i][j]*globs.myDxx[i][2] )
                                * myResult[k][i+1][j];
                }

                //  explicit y ; RAW v, write u
                v[k][i][j] = 0.0;

                if(j > 0) {
                  v[k][i][j] +=  ( 0.5*myVarY[i][j]*globs.myDyy[j][0] )
                             *  myResult[k][i][j-1];
                }
                v[k][i][j]  +=   ( 0.5*myVarY[i][j]*globs.myDyy[j][1] )
                             *  myResult[k][i][j];
                if(j < numY-1) {
                  v[k][i][j] +=  ( 0.5*myVarY[i][j]*globs.myDyy[j][2] )
                             *  myResult[k][i][j+1];
                }
                u[k][j][i] += v[k][i][j]; 

                
                //  implicit x  // write a,b,c
                a[k][j][i] =       - 0.5*(0.5*myVarX[i][j]*globs.myDxx[i][0]);
                b[k][j][i] = ( 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g])) - 0.5*(0.5*myVarX[i][j]*globs.myDxx[i][1]);
                c[k][j][i] =       - 0.5*(0.5*myVarX[i][j]*globs.myDxx[i][2]);
            }
        }
    }

#pragma omp parallel for default(shared) schedule(static)   // Kernel-3: 3D
    for( unsigned k = 0; k < outer; ++ k ) {  //outermost loop distribution //Par
            for(unsigned j=0;j<numY;j++) {  // Par
                tridagPar(a[k][j],b[k][j],c[k][j],u[k][j],numX,u[k][j],yy[k][j]);  
            }
        }

    bool valid = true;

    for(int iter = 0; iter < outer*numY*numX; iter ++){
        unsigned k = iter /numY/numX;
        unsigned j = iter/ numX;
        unsigned i = iter % numX;
        if(i< numX && j < numY){
            // REAL temp_a = - 0.5*(0.5*myVarX[i][j]*globs.myDxx[i][0]);
            // if(abs(h_a[iter] - a[k][j][i]) > EPSILON ||
            //     abs(h_b[iter] - b[k][j][i]) > EPSILON || 
            //     abs(h_c[iter] - c[k][j][i]) > EPSILON){

            //     valid = false;
            //     printf("\n** [h_a]  k %d,  j %d, i %d, :  %f != %f **\n",
            //         k,j,i, h_a[iter], a[k][j][i]);
            //     printf("\n** [h_b] did not validate ! k %d,  j %d, i %d, :  %f != %f **\n",
            //         k,j,i, h_b[iter], b[k][j][i]);
            //     printf("\n** [h_c] did not validate ! k %d,  j %d, i %d, :  %f != %f **\n",
            //         k,j,i, h_c[iter], c[k][j][i]);

            // }  
            if (abs(h_u[iter] - u[k][j][i]) > EPSILON ) > EPSILON ) {
                valid = false;
                printf("\n** [h_u] did not validate ! k %d,  j %d, i %d, :  %f != %f **\n",
                    k,j,i, h_u[iter], u[k][j][i]);
                break;
            }
        }
    }

   
    if(!valid){
        printf("\n** GPU_ROLLBACK_PART_2_TEST did not validate**\n");
    }

    free(h_a); free(h_b);free(h_c);
    // free(h_yy);free(h_yyy);
    free(h_u); free(h_v);
#endif






    hipFree(d_x); hipFree(d_y); hipFree(d_dxx);hipFree(d_dyy); hipFree(d_timeline); 
    hipFree(d_result); hipFree(d_varX); hipFree(d_varY);
    hipFree(d_a); hipFree(d_b);hipFree(d_c); hipFree(d_yy);hipFree(d_yyy); 
    hipFree(d_u); hipFree(d_v);
    free(h_result);

    //SHould perhaps be initialized on the gpu instead to save PCI bandwidth. Possibly negl
    /*
     * setPayOff: 
     * INPUT: globs.myX
     * Output: myResult
     *
     * updateParams:
     * input: globs.myTimeline, globs.myX, globs.myY, alpha, beta,
     * output: myVarX, myVarY
     *
     * rollback-1:
     * input: globs.myTimeLine, myResult, 
     * output: 
     *
     * tridagPar:
     *
     * rollback-2:
     * input:
     * output:
     * */

 //   #endif
}

